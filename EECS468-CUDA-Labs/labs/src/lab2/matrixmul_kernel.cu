#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

#define TILE_WIDTH 16

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	__shared__ float subM[TILE_WIDTH][TILE_WIDTH];
	__shared__ float subN[TILE_WIDTH][TILE_WIDTH];

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int by = blockIdx.y;
	const int bx = blockIdx.x;

	const int j = M.height; // or P.height
	const int k = M.width; // or N.height
	const int l = N.width; // or P.width

	int row = by*TILE_WIDTH+ty;
	int col = bx*TILE_WIDTH+tx;

	float pValue = 0.0f;

	for (int i = 0; i < (int)ceil(k/(float)TILE_WIDTH); ++i) {

		if ((row<j) && (i*TILE_WIDTH+tx) < k) {
			subM[ty][tx] = M.elements[row*k + i*TILE_WIDTH + tx];
		} else {
			subM[ty][tx] = 0.0f;
		}

		if ((i*TILE_WIDTH+ty) < k && col<l) {
			subN[ty][tx] = N.elements[(i*TILE_WIDTH+ty) * l + col];
		} else {
			subN[ty][tx] = 0.0f;
		}

		__syncthreads();

		for (int idx = 0; idx < TILE_WIDTH; ++idx) {
			pValue += subM[ty][idx] * subN[idx][tx];
		}

		__syncthreads();
	}

	if ((row<j) && (col<l)) {
		P.elements[row * l + col] = pValue;
	}
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
