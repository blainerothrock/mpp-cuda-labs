#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"
#include "opt_2dhisto.h"

__global__ void opt_2dhisto_kernel(uint32_t *input, size_t *inputHeight, size_t *inputWidth, uint32_t bins[HISTO_HEIGHT*HISTO_WIDTH])
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int numThreads = blockIdx.x * blockDim.x;
	const int binSize = HISTO_HEIGHT*HISTO_WIDTH;

	__shared__ uint32_t sBins[binSize];

	for ( int pos = threadIdx.x; pos < binSize; pos += blockDim.x )
		sBins[pos] = 0;

	__syncthreads();

	int binIdx = input[tid];
	atomicAdd(&sBins[binIdx], 1);

	__syncthreads();

	for ( int pos = threadIdx.x; pos < binSize; pos += blockDim.x ) {
		atomicAdd(&bins[pos], sBins[pos]);
	}
}

uint32_t * allocCopyInput(uint32_t **input, size_t width, size_t height)
{
    // solution from http://www.trevorsimonton.com/blog/2016/11/16/transfer-2d-array-memory-to-cuda.html
    uint32_t** flattenedRepresentation = new uint32_t*[height];
    flattenedRepresentation[0] = new uint32_t[height * width];
    for (int i = 1; i < height; ++i) flattenedRepresentation[i] = flattenedRepresentation[i-1] + width;

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            flattenedRepresentation[i][j] = input[i][j];
        }
    }

    uint32_t *input_d;
//    uint32_t *input_device;
    int sizeInput = width*height*sizeof(uint32_t);
    hipError_t allocError = hipMalloc((void **)&input_d, sizeInput);
    //printf("input alloc error: %s\n", hipGetErrorString(allocError));
    hipError_t cpyError = hipMemcpy(input_d, flattenedRepresentation[0], sizeInput, hipMemcpyHostToDevice);
    //delete [] flattenedRepresentation;
    //printf("input cpy error: %s\n", hipGetErrorString(cpyError));
    return input_d;
}

uint32_t * allocCopyBin()
{
    uint32_t *bins_d;
    int sizeBins = HISTO_HEIGHT*HISTO_WIDTH*sizeof(uint32_t);
    hipError_t allocError = hipMalloc((void **)&bins_d, sizeBins);
    //printf("bin alloc error: %s\n", hipGetErrorString(allocError));
//    hipError_t cpyError = hipMemcpy(bins_d, bins, sizeBins, hipMemcpyHostToDevice);
//    printf("bin cpy error: %s\n", hipGetErrorString(cpyError));
    hipError_t memSetError = hipMemset(bins_d, 0, sizeBins);
    //printf("bin mem set error: %s\n", hipGetErrorString(memSetError));
    return bins_d;
}

size_t * allocCopyDim(size_t inputDim)
{
    size_t *inputDim_d;
    hipError_t allocError = hipMalloc((void **) &inputDim_d, sizeof(size_t));
    //printf("dim alloc error: %s\n", hipGetErrorString(allocError));
    hipError_t cpyError = hipMemcpy(inputDim_d, &inputDim, sizeof(size_t), hipMemcpyHostToDevice);
    //printf("dim cpy error: %s\n", hipGetErrorString(cpyError));
    return inputDim_d;
}

void copyBinsFromDevice(uint8_t h_bins[HISTO_HEIGHT*HISTO_WIDTH], uint32_t d_bins[HISTO_HEIGHT*HISTO_WIDTH]){
	uint32_t tmpBins[HISTO_HEIGHT*HISTO_WIDTH];

	int sizeTmpBins = HISTO_HEIGHT*HISTO_WIDTH*sizeof(uint32_t);
	hipError_t cpyError = hipMemcpy(tmpBins, d_bins, sizeTmpBins, hipMemcpyDeviceToHost);
    //printf("Copy D to H error: %s\n", hipGetErrorString(cpyError));

    for ( int i = 0; i < HISTO_HEIGHT*HISTO_WIDTH; i++ ) {
    	if (tmpBins[i] > 255) tmpBins[i] = 255;
    	h_bins[i] = static_cast<uint8_t>(tmpBins[i]);
    }
}

void freeMemory(uint32_t *input, size_t *height, size_t *width, uint32_t bins[HISTO_HEIGHT*HISTO_WIDTH] ){
	//printf("Freeing memory\n");
	hipFree(input);
	input = NULL;
	hipFree(height);
	height = NULL;
	hipFree(width);
	width = NULL;
	hipFree(bins);
	bins = NULL;
}


void opt_2dhisto( uint32_t *input, size_t *height, size_t *width, uint32_t bins[HISTO_HEIGHT*HISTO_WIDTH] )
{
    //dim3 DimGrid(31872, 1);
    float numThreads = 1024.0;
    float inputSize = INPUT_HEIGHT * INPUT_WIDTH;
    float numBlocks = ceilf(inputSize / numThreads);

    // set the bins count to 0
    hipMemset(bins, 0, HISTO_HEIGHT*HISTO_WIDTH);

//    printf("\nprinting first 5 values of bins:");
//    printf(bins[0]);
//    for(int i = 0; i < 5; i++){
//        printf("\t %i", bins[i]);
//        printf("\n");
//    }

//    unsigned int BIN_COUNT= HISTO_HEIGHT*HISTO_WIDTH;
    opt_2dhisto_kernel<<<numBlocks,numThreads>>>(input, height, width, bins);

    hipDeviceSynchronize();
    hipError_t error;
    error = hipGetLastError();
    //printf("error: %s\n", hipGetErrorString(error));

}

/* Include below the implementation of any other functions you need */ //



