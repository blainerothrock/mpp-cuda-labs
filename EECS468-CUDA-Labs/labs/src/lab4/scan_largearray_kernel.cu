#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>

#define numThreads 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 512

// Lab4: Host Helper Functions (allocate your own data structure...)


// Lab4: Device Functions


// Lab4: Kernel Functions

__global__ void prescanKernel(float *outArray, float *inArray, int numElements){

	// scan arr in shared mem
	extern __shared__ float scanArray[];

	//printf("threadidx.x: %i\n", threadIdx.x);

	scanArray[threadIdx.x] = (threadIdx.x > 0) ? inArray[blockIdx.x * numThreads + threadIdx.x - 1] : 0;


	__syncthreads();
	// exclusive

    // reduction step
	int stride = 1;

	while (stride < numThreads){
        __syncthreads();
		int index = (threadIdx.x + 1) * stride * 2;

		if (index < numThreads)
			scanArray[index] += scanArray[index-stride];

		stride *= 2;

	}


	// post-scan step
	stride = numThreads >> 1;

	while (stride > 0){
        __syncthreads();
		int index = (threadIdx.x + 1) * stride * 2;

		if (index < numThreads)
			scanArray[index+stride] += scanArray[index];

		stride = stride >> 1;

	}


    __syncthreads();
	outArray[blockIdx.x * numThreads + threadIdx.x] = scanArray[threadIdx.x];

}

// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	printf("smem size: %i\n", numThreads * sizeof(float));

//    dim3 DimGrid(1,1);
    printf("num elements: %i\n", numElements);

    //TODO:Make dynamic for numElements not divisible by 1024 (Multiple and Remainder)
    const int numBlocks = numElements/numThreads;
    dim3 DimBlock(numBlocks);
    int sharedMemSize = numThreads * sizeof(float);

    prescanKernel<<<DimBlock, numThreads,sharedMemSize>>>(outArray, inArray, numElements);

}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
