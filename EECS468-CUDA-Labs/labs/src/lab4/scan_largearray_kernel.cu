#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>

#define numThreads 4
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 512

// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions


// Lab4: Kernel Functions

__device__ float blockSums[4];

__global__ void prescanKernel(float *outArray, float *inArray, int numElements){

	// scan arr in shared mem
	extern __shared__ float scanArray[];

	//printf("threadidx.x: %i\n", threadIdx.x);
	if (blockIdx.x == 0 && threadIdx.x == 0)
		scanArray[threadIdx.x] = 0;
	else
		scanArray[threadIdx.x] = inArray[blockIdx.x * numThreads + threadIdx.x - 1];


	__syncthreads();
	// exclusive

	if (threadIdx.x == numThreads-1)
		printf("Block Id: %i, scanArray before scan!: %.1f %.1f %.1f %.1f\n", blockIdx.x, scanArray[0],scanArray[1], scanArray[2], scanArray[3]);

    // reduction step
	int stride = 1;

	while (stride < numThreads){
		int index;
		// exclusive for the first block
		if (blockIdx.x == 0)
			index = (threadIdx.x + 1) * stride * 2;
		// inclusive for remaining blocks
		else
			index = (threadIdx.x + 1) * stride * 2 - 1;

		if (index < numThreads)
			scanArray[index] += scanArray[index-stride];

		stride *= 2;

		__syncthreads();
	}

	// post-scan step
	stride = numThreads >> 1;

	while (stride > 0){
		// don't need to check block, inclusive
		int index;
		if (blockIdx.x == 0)
			index = (threadIdx.x + 1) * stride * 2;
		else
			index = (threadIdx.x + 1) * stride * 2 - 1;

		if (index < numThreads)
			scanArray[index+stride] += scanArray[index];

		stride = stride >> 1;

		__syncthreads();

	}

	__syncthreads();

	//int index = (threadIdx.x + 1) * stride * 2;


	if (threadIdx.x == 0) {
//		if (blockIdx.x < 1) {
//			printf("setting %i to %f\n", blockIdx.x, scanArray[numThreads - 1]);
//		}
		printf("Block Id: %i, scanArray after scan!:  %.1f %.1f %.1f %.1f\n", blockIdx.x, scanArray[0],scanArray[1], scanArray[2], scanArray[3]);
		blockSums[blockIdx.x] = scanArray[numThreads - 1];
	}

    __syncthreads();


	outArray[blockIdx.x * numThreads + threadIdx.x] = scanArray[threadIdx.x];

}

__global__ void blockScanKernel(float *outArray) {

	//printf("\nHello from Second Kernel. Printing blockSums");
	//

	extern __shared__ float scanArray[];
	int numT = blockDim.x;

	if (blockIdx.x == 0 && threadIdx.x == 0)
		scanArray[threadIdx.x] = 0;
	else
		scanArray[threadIdx.x] = blockSums[blockIdx.x * blockDim.x + threadIdx.x - 1];

	__syncthreads();
	// exclusive

	if (threadIdx.x == numT-1)
		printf("Block Id: %i, 2nd scanArray before scan!: %.1f %.1f\n", blockIdx.x, scanArray[0],scanArray[1]);

	// reduction step
	int stride = 1;

	while (stride < numT){
		__syncthreads();
		int index = (threadIdx.x + 1) * stride * 2;

		if (index < numT)
			scanArray[index] += scanArray[index-stride];

		stride *= 2;
	}

	// post-scan step
	stride = numT >> 1;

	while (stride > 0){
		__syncthreads();
		int index = (threadIdx.x + 1) * stride * 2;

		if (index < numT)
			scanArray[index+stride] += scanArray[index];

		stride = stride >> 1;

	}

//	__syncthreads();
//
//	blockSums[blockIdx.x * numThreads + threadIdx.x] = scanArray[threadIdx.x];
	__syncthreads();


	//printf("\nBlockSums %f %f %f %f", blockSums[0],blockSums[1], blockSums[2], blockSums[3]);

	// print out scanArray (should be the scan of blockSums, step 3)
	if (threadIdx.x == 0)
		printf("BlockSums: %.1f %.1f %.1f %.1f  ----- 2nd scanArray after scan!: %.1f %.1f \n", blockSums[0], blockSums[1], blockSums[2], blockSums[3], scanArray[0], scanArray[1]);

	// add to out array
	outArray[blockIdx.x * numT + threadIdx.x] += blockSums[blockIdx.x];

}

// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{
//    dim3 DimGrid(1,1);
    printf("num elements: %i\n", numElements);

    //TODO: Make dynamic for numElements not divisible by 1024 (Multiple and Remainder)
    const int numBlocks = numElements/numThreads;
    dim3 DimBlock(numBlocks);
    int sharedMemSize = numThreads * sizeof(float);

    printf("smem size: %i\n", sharedMemSize);

    prescanKernel<<<DimBlock, numThreads,sharedMemSize>>>(outArray, inArray, numElements);

    const int numBlocks1 = 2;
    const int numThreads1 = 2;
    dim3 DimBlock1(numBlocks1);
    blockScanKernel<<<DimBlock1, numThreads1,sharedMemSize>>>(outArray);

}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
